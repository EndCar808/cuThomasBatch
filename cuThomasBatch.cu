#include "hip/hip_runtime.h"
// Enda Carroll
// Sept 2019
// Function declarations for cuThomasBatch routine to solve batches of tridiagonal systems

//   Copyright 2019 Enda Carroll

//   Licensed under the Apache License, Version 2.0 (the "License");
//   you may not use this file except in compliance with the License.
//   You may obtain a copy of the License at

//       http://www.apache.org/licenses/LICENSE-2.0

//   Unless required by applicable law or agreed to in writing, software
//   distributed under the License is distributed on an "AS IS" BASIS,
//   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//   See the License for the specific language governing permissions and
//   limitations under the License.


// ---------------------------------------------------------------------
//  Standard Libraries and Headers
// ---------------------------------------------------------------------



// ---------------------------------------------------------------------
//  User Libraries and Headers
// ---------------------------------------------------------------------

#include "cuThomasBatch.h"



/**
* Function to perform a prefactorization of the LHS using the Thomas algorithm (performed on host device)
*
* @param la Lower diagonal of the LHS matrix - array of lenght n
* @param lb Main diagonal of the LHS matrix - array of lenght n
* * @param lc Upper diagonal of the LHS matrix - array of lenght n
* @param n  Size of the system being solved
*/
void thomasFactorConstantBatch(double* la, double* lb, double* lc, int n) {

	int rowCurrent;
	int rowPrevious;

	rowCurrent = 0;

	// First row
	lb[rowCurrent] = lb[rowCurrent];
	lc[rowCurrent] = lc[rowCurrent] / lb[rowCurrent];

	for (int i = 1; i < n - 1; ++i)	{
		rowPrevious = rowCurrent;
		rowCurrent  += 1;

		la[rowCurrent] = la[rowCurrent];
		lb[rowCurrent] = lb[rowCurrent] - la[rowCurrent]*lc[rowPrevious];
		lc[rowCurrent] = lc[rowCurrent] / lb[rowCurrent];
	}

	rowPrevious = rowCurrent;
	rowCurrent += 1;

	// Last row
	la[rowCurrent] = la[rowCurrent];
	lb[rowCurrent] = lb[rowCurrent] - la[rowCurrent]*lc[rowPrevious];
}

/**
* Kernel to solve a prefactorized system using the Thomas alogrithm
* 
* @param la Lower diagonal of the LHS matrix - array of lenght n
* @param lb Main diagonal of the LHS matrix - array of lenght n
* @param lc Upper diagonal of the LHS matrix - array of lenght n
* @param d  RHS array - size n by m
* @param n  Size of the system being solved
* @param m  Size of the batch 
*/
__global__ void cuThomasBatch(double* la, double* lb, double* lc, double* d, int n, int m ) {

	int rowCurrent;
	int rowPrevious;

	int rowAhead;

	// set the current row
	rowCurrent = threadIdx.x + blockDim.x*blockIdx.x;

	int i = 0;

	if ( rowCurrent < m ) 
	{

		//----- Forward Sweep
		d[rowCurrent] = d[rowCurrent] / lb[i];

		#pragma unroll
		for (i = 1; i < n; ++i) {
			rowPrevious = rowCurrent;
			rowCurrent += m;

			d[rowCurrent] = (d[rowCurrent] - la[i]*d[rowPrevious]) / (lb[i]);
		
		}


		//----- Back Sub
		d[rowCurrent] = d[rowCurrent];

		#pragma unroll
		for (i = n - 2; i >= 0; --i) {
			rowAhead    = rowCurrent;
			rowCurrent -= m;

			d[rowCurrent] = d[rowCurrent] - lc[i] * d[rowAhead];
		}
	}
}